#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int main() {
    int numRowsX = 1024;
    int numColsX = 512;
    int numColsY = 2048;

    float* hostX = (float*)malloc(numRowsX * numColsX * sizeof(float));
    float* hostY = (float*)malloc(numColsX * numColsY * sizeof(float));
    float* hostZ = (float*)malloc(numRowsX * numColsY * sizeof(float));

    srand(time(NULL));
    for (int i = 0; i < numRowsX * numColsX; i++) {
        hostX[i] = rand() / (float)RAND_MAX;
    }
    for (int i = 0; i < numColsX * numColsY; i++) {
        hostY[i] = rand() / (float)RAND_MAX;
    }

    float* deviceX, * deviceY, * deviceZ;
    hipMalloc((void**)&deviceX, numRowsX * numColsX * sizeof(float));
    hipMalloc((void**)&deviceY, numColsX * numColsY * sizeof(float));
    hipMalloc((void**)&deviceZ, numRowsX * numColsY * sizeof(float));

    hipMemcpy(deviceX, hostX, numRowsX * numColsX * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceY, hostY, numColsX * numColsY * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 16;
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 numBlocks(ceil(numColsY / (float)blockSize), ceil(numRowsX / (float)blockSize));

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    MultiplyMatrix<<<numBlocks, threadsPerBlock>>>(deviceX, deviceY, deviceZ, numRowsX, numColsX, numColsY);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(hostZ, deviceZ, numRowsX * numColsY * sizeof(float), hipMemcpyDeviceToHost);

    printf("Elapsed time: %f ms\n", elapsedTime);

    free(hostX);
    free(hostY);
    free(hostZ);
    hipFree(deviceX);
    hipFree(deviceY);
    hipFree(deviceZ);

    return 0;
}
void printMatrix(float* matrix, int numRows, int numCols) {
    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numCols; j++) {
            printf("%f ", matrix[i * numCols + j]);
        }
        printf("\n");
    }
}
__global__ void MultiplyMatrix(float* X, float* Y, float* Z, int h, int w, int d)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < h && col < d) {
        float output = 0;
        for (int k = 0; k < w; k++) {
            output +=X[row * w + k] * Y[k * d + col];
        }
        Z[row* d + col] = output;
    }
}